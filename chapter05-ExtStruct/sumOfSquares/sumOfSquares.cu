#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application, doesn't use cutil library.
*/

#include <stdio.h>
#include <string.h>
#include <iostream>
using namespace std;

#include <hip/hip_runtime.h>

#include <cutil.h>
#include <cutil_inline_runtime.h>   

#include "modifyVertexByJoint.cuh"
#include "StructMS3D.h"

#define DATA_SIZE		(1<<13)//8k ����Ƭ
//#define DATA_SIZE		((1<<10)*24)//24k ����
#define ATTRIB_SIZE		(1<<6)	//64 ������

//#define THREAD_NUM  (1<<7)//128
//#define BLOCK_NUM    ((1<<4)*6)//96

#define TIMES_REPERT	(1<<0)

int THREAD_NUM, BLOCK_NUM;

Ms3dVertexArrayElement pVertexArray[DATA_SIZE*3];
Ms3dVertexArrayElement pVertexArrayBackup[DATA_SIZE*3];
DMs3dJoint	pJoints[ATTRIB_SIZE];

// ��ʱ�����ο���http://soulshengbbs.sinaapp.com/thread-12-1-1.html ��cuda����ʱ��ķ������ܡ�����cutGetTimerValue
unsigned int hTimer ;
void timeBegin()
{
	cutilDeviceSynchronize() ;
	cutStartTimer(hTimer) ;
	cutResetTimer(hTimer);
}
void timeEnd(string msg)
{
	cutilDeviceSynchronize() ;
	cutStopTimer(hTimer) ;

	double Passed_Time = cutGetTimerValue(hTimer);

	printf("time��%s��: %.3f ms\n", msg.c_str(), Passed_Time);
}

clock_t clockBegin,clockEnd;
void timeBeginCPU()
{
	clockBegin = clock();
}
void timeEndCPU(string msg)
{
	clockEnd = clock();

	double Passed_Time = clockEnd - clockBegin;

	printf("time��%s��: %.3f ms\n", msg.c_str(), Passed_Time);
}

bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

	int i;
    for( i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);
	
    printf("CUDA initialized.\n\n");
    return true;
}



void GenerateNumbers(Ms3dVertexArrayElement *pVert, int size)
{
	float *pVertFloat = (float *)pVert;
    for(int i = 0; i < size * sizeof(Ms3dVertexArrayElement)/sizeof(float); i++) {
        pVertFloat[i] = rand() % 100;
    }
}

__device__
void deviceTransformVetex(float* pos, float* mat)
{
	float* m_fMat = mat;

	float x = pos[0] * m_fMat[0] + 
		pos[1] * m_fMat[4] +
		pos[2] * m_fMat[8] + 
		m_fMat[12] ;

	float y = pos[0] * m_fMat[1] + 
		pos[1] * m_fMat[5] + 
		pos[2] * m_fMat[9] + 
		m_fMat[13] ;

	float z = pos[0] * m_fMat[2] + 
		pos[1] * m_fMat[6] + 
		pos[2] * m_fMat[10]+
		m_fMat[14] ;

	pos[0] = x;
	pos[1] = y;
	pos[2] = z;

}
__global__ void modifyVertexByJointInGPUKernel( float* pVertexArray, float* pVertexArrayBackup, 
	DMs3dJoint * pJoints, int nTriangleIndices , clock_t* time )
{
	int loop1 = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

	if ( loop1 >= nTriangleIndices )
	{
		return;
	}

	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	if(tid == 0) time[bid] = clock();
	
	// ������������������� 
	for(int z = 0; z < 3; z++)
	{
		int vertexCnt = 9 * (loop1 * 3 + z);

		Ms3dVertexArrayElement* pVert = (Ms3dVertexArrayElement*)(pVertexArray + vertexCnt);
		Ms3dVertexArrayElement* pVertBackup = (Ms3dVertexArrayElement*)(pVertexArrayBackup + vertexCnt);
		
		pVert->m_vVert[0] = pVertBackup->m_vVert[0] ;
		pVert->m_vVert[1] = pVertBackup->m_vVert[1] ;
		pVert->m_vVert[2] = pVertBackup->m_vVert[2] ;

		int nIdBone = (int)( pVertBackup->m_fBone + 0.5f );
		if(nIdBone != -1)
		{			
			DMs3dJoint * pJoint = &pJoints[ nIdBone];

			deviceTransformVetex( pVert->m_vVert, pJoint->m_matFinal );
		}
	}

	if(tid == 0) time[bid + gridDim.x] = clock();

}
#if 0
__global__ static void sumOfSquares(int *num, int* result, clock_t* time)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	
	int sum ;
	int i;
	if(tid == 0) time[bid] = clock();
	for(i = tid + bid * THREAD_NUM; i < DATA_SIZE; i+= THREAD_NUM * BLOCK_NUM) {
		sum += num[i] * num[i] ;
	}

	result[tid + bid * THREAD_NUM] = sum;
    if(tid == 0) time[bid + BLOCK_NUM] = clock();
}
#endif
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

void runCUDA()
{
	void *dVert, *dVertBackup, *dJoint;
	clock_t* time;
   computeGridSize(DATA_SIZE, 256, BLOCK_NUM, THREAD_NUM);

	hipMalloc((void**) &time, sizeof(clock_t) * BLOCK_NUM * 2);
	hipMalloc((void**) &dVert, sizeof(Ms3dVertexArrayElement) * DATA_SIZE*3);
	hipMalloc((void**) &dVertBackup, sizeof(Ms3dVertexArrayElement) * DATA_SIZE*3);
	hipMalloc((void**) &dJoint, sizeof(DMs3dJoint) * ATTRIB_SIZE);
	hipMemcpy(dVert, pVertexArray, sizeof(Ms3dVertexArrayElement) * DATA_SIZE*3, hipMemcpyHostToDevice);
	hipMemcpy(dVertBackup, pVertexArrayBackup, sizeof(Ms3dVertexArrayElement) * DATA_SIZE*3, hipMemcpyHostToDevice);
	hipMemcpy(dJoint, pJoints, sizeof(DMs3dJoint) * ATTRIB_SIZE, hipMemcpyHostToDevice);

	//sumOfSquares<<<BLOCK_NUM, THREAD_NUM, 0>>>(gpudata, result, time);
	modifyVertexByJointInGPUKernel<<< BLOCK_NUM, THREAD_NUM >>>
		( (float*)pVertexArray, (float*)pVertexArrayBackup, pJoints, DATA_SIZE, time);

	
    clock_t *time_used = new clock_t[BLOCK_NUM * 2];
    hipMemcpy(time_used, time, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost);
	hipFree(dVert);
	hipFree(dVertBackup);
	hipFree(dJoint);

	clock_t min_start, max_end;
    min_start = time_used[0];
    max_end = time_used[BLOCK_NUM];
    for(int i = 1; i < BLOCK_NUM; i++) {
        if(min_start > time_used[i])
            min_start = time_used[i];
        if(max_end < time_used[i + BLOCK_NUM])
            max_end = time_used[i + BLOCK_NUM];
    }
	delete []time_used;
    printf("time: %d, time/n: %.4f\n", max_end - min_start, (max_end - min_start)*1.0f/DATA_SIZE);
}

void runCPU()
{
#if 0
	int final_sum = 0;
	 for(int i = 0; i < DATA_SIZE; i++) {
		 final_sum += data[i] * data[i];
	 }
	//printf("sum��CPU��: %d of %d squares\n", final_sum, DATA_SIZE);
#endif
}

int main(int argc, char **argv)
{
	// ��ʼ��cuda
	if(!InitCUDA()) {
		return 0;
	}

	// ���ݳ�ʼ��
	GenerateNumbers(pVertexArray, DATA_SIZE*3);
	GenerateNumbers(pVertexArrayBackup, DATA_SIZE*3);

	cutCreateTimer(&hTimer);

	timeBegin();
	// cuda����
	runCUDA();
	timeEnd("cuda");
	
	// cpu����
	runCPU();


	cutDeleteTimer(hTimer);

	//system("pause");
}

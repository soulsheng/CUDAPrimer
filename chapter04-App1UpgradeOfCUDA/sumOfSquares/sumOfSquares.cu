#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application, doesn't use cutil library.
*/

#include <stdio.h>
#include <string.h>
#include <iostream>

//#include <shrQATest.h>
#include <hip/hip_runtime.h>

using namespace std;

bool g_bQATest = false;

#ifndef MIN
#define MIN(a,b) ((a < b) ? a : b)
#endif
#ifndef MAX
#define MAX(a,b) ((a > b) ? a : b)
#endif

#define DATA_SIZE (1<<20)//1048576
#define THREAD_NUM   256
#define BLOCK_NUM   32
int data[DATA_SIZE];

#ifdef _WIN32
   #define STRCASECMP  _stricmp
   #define STRNCASECMP _strnicmp
#else
   #define STRCASECMP  strcasecmp
   #define STRNCASECMP strncasecmp
#endif

#define ASSERT(x, msg, retcode) \
    if (!(x)) \
    { \
        cout << msg << " " << __FILE__ << ":" << __LINE__ << endl; \
        return retcode; \
    }

__global__ void sequence_gpu(int *d_ptr, int length)
{
    int elemID = blockIdx.x * blockDim.x + threadIdx.x;
    if (elemID < length)
    {
        d_ptr[elemID] = elemID;
    }
}


void sequence_cpu(int *h_ptr, int length)
{
    for (int elemID=0; elemID<length; elemID++)
    {
        h_ptr[elemID] = elemID;
    }
}

void processArgs(int argc, char **argv)
{
    for (int i=1; i < argc; i++) {
        if((!STRNCASECMP((argv[i]+1), "noprompt", 8)) || (!STRNCASECMP((argv[i]+2), "noprompt", 8)) )
        {
            g_bQATest = true;
        }
    }
}
// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct {
       int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
       int Cores;
    } sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = 
	{ { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
	  { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
	  { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
	  { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
	  { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
	  { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
	  { 0x30, 192}, // Fermi Generation (SM 3.0) GK10x class
	  {   -1, -1 }
	};

    int index = 0;
    while (nGpuArchCoresPerSM[index].SM != -1) {
       if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
          return nGpuArchCoresPerSM[index].Cores;
       }	
       index++;
    }
    printf("MapSMtoCores undefined SM %d.%d is undefined (please update to the latest SDK)!\n", major, minor);
    return -1;
}
// Ӳ��ӵ�����ĸ����������GFLOPS
int gpuGetMaxGflopsDeviceId(float& fGFLOPS)
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount( &device_count );
    
    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999)
        {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
    }

    // Find the best CUDA capable GPU device
    current_device = 0;
	while( current_device < device_count )
	{
		hipGetDeviceProperties( &deviceProp, current_device );
		if (deviceProp.major == 9999 && deviceProp.minor == 9999)
		{
			sm_per_multiproc = 1;
		}
		else
		{
			sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
		}

		int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
		// clockRateָshader��Ƶ�ʣ���λ��kHz����"Clock frequency in kilohertz "���ο���http://developer.download.nvidia.com/compute/cuda/4_2/rel/toolkit/docs/online/structcudaDeviceProp_dee14230e417cb3059d697d6804da414.html#dee14230e417cb3059d697d6804da414

		if( compute_perf  > max_compute_perf )
		{
			// If we find GPU with SM major > 2, search only these
			if ( best_SM_arch > 2 )
			{
				// If our device==dest_SM_arch, choose this, or else pass
				if (deviceProp.major == best_SM_arch)
				{
					max_compute_perf  = compute_perf;
					max_perf_device   = current_device;
				}
			}
			else
			{
				max_compute_perf  = compute_perf;
				max_perf_device   = current_device;
				printf("sp������%d, shaderƵ��: %d \n", deviceProp.multiProcessorCount * sm_per_multiproc, deviceProp.clockRate);
			}
		}
		++current_device;
	}
	fGFLOPS = max_compute_perf * 1.0e-6;
	return max_perf_device;
}
bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
#if 1
	float fGFLOPS = 0.0f;
	i = gpuGetMaxGflopsDeviceId( fGFLOPS );
    printf("�����������㹫ʽ=sp���� * shaderƵ�� \n\
			�����������Թ���: %0.2f GFLOPS\n", fGFLOPS);
#else
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }
#endif
    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}



void GenerateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}

__global__ static void sumOfSquares(int *num, int* result, clock_t* time)
{
	extern __shared__ int shared[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

    int sum = 0;
    int i;
	
	if(tid == 0) time[bid] = clock();
	shared[tid] = 0;
    for(i = tid + bid * THREAD_NUM; i < DATA_SIZE; i+= THREAD_NUM * BLOCK_NUM) {
        shared[tid] += num[i] * num[i];
    }

	__syncthreads();

	if(tid == 0){
		for(i=1;i< THREAD_NUM;i++)
			shared[0] += shared[i];
		result[bid] = shared[0];
	}

    if(tid == 0) time[bid + BLOCK_NUM] = clock();
}

int main(int argc, char **argv)
{
	//shrQAStart(argc, argv);
	 if(!InitCUDA()) {
        return 0;
    }

    printf("CUDA initialized.\n");

	 GenerateNumbers(data, DATA_SIZE);

    int* gpudata, *result;
	clock_t* time;
    hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &result, sizeof(int) * BLOCK_NUM);
    hipMalloc((void**) &time, sizeof(clock_t) * BLOCK_NUM * 2);
    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

	sumOfSquares<<<BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int)>>>(gpudata, result, time);

    int sum[BLOCK_NUM];
    clock_t time_used[BLOCK_NUM * 2];
    hipMemcpy(&sum, result, sizeof(int) * BLOCK_NUM , hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);

	int final_sum = 0;
    for(int i = 0; i < BLOCK_NUM; i++) {
        final_sum += sum[i] ;
    }
    printf("sum��GPU��: %d\n", final_sum);

	clock_t min_start, max_end;
    min_start = time_used[0];
    max_end = time_used[BLOCK_NUM];
    for(int i = 1; i < BLOCK_NUM; i++) {
        if(min_start > time_used[i])
            min_start = time_used[i];
        if(max_end < time_used[i + BLOCK_NUM])
            max_end = time_used[i + BLOCK_NUM];
    }
    printf("time: %d, time/n: %.2f\n", max_end - min_start, (max_end - min_start)*1.0f/DATA_SIZE);

	final_sum = 0;
    for(int i = 0; i < DATA_SIZE; i++) {
        final_sum += data[i] * data[i];
    }
    printf("sum��CPU��: %d of %d squares\n", final_sum, DATA_SIZE);

#if 0
    cout << "CUDA Runtime API template" << endl;
    cout << "=========================" << endl;
    cout << "Self-test started" << endl;

    const int N = 100;

    processArgs(argc, argv);

    int *d_ptr;
    ASSERT(hipSuccess == hipMalloc    (&d_ptr, N * sizeof(int)), "Device allocation of " << N << " ints failed", -1);

    int *h_ptr;
    ASSERT(hipSuccess == hipHostMalloc(&h_ptr, N * sizeof(int)), "Host allocation of "   << N << " ints failed", -1);

    cout << "Memory allocated successfully" << endl;

    dim3 cudaBlockSize(32,1,1);
    dim3 cudaGridSize((N + cudaBlockSize.x - 1) / cudaBlockSize.x, 1, 1);
    sequence_gpu<<<cudaGridSize, cudaBlockSize>>>(d_ptr, N);
    ASSERT(hipSuccess == hipGetLastError(), "Kernel launch failed", -1);
    ASSERT(hipSuccess == hipDeviceSynchronize(), "Kernel synchronization failed", -1);

    sequence_cpu(h_ptr, N);

    cout << "CUDA and CPU algorithm implementations finished" << endl;

    int *h_d_ptr;
    ASSERT(hipSuccess == hipHostMalloc(&h_d_ptr, N * sizeof(int)), "Host allocation of " << N << " ints failed", -1);
    ASSERT(hipSuccess == hipMemcpy(h_d_ptr, d_ptr, N * sizeof(int), hipMemcpyDeviceToHost), "Copy of " << N << " ints from device to host failed", -1);
    bool bValid = true;
    for (int i=0; i<N && bValid; i++)
    {
        if (h_ptr[i] != h_d_ptr[i])
        {
            bValid = false;
        }
    }

    ASSERT(hipSuccess == hipFree(d_ptr),       "Device deallocation failed", -1);
    ASSERT(hipSuccess == hipHostFree(h_ptr),   "Host deallocation failed",   -1);
    ASSERT(hipSuccess == hipHostFree(h_d_ptr), "Host deallocation failed",   -1);

    cout << "Memory deallocated successfully" << endl;
    cout << "TEST Results " << endl;
#endif  
    //shrQAFinishExit(argc, (const char **)argv, (bValid ? QA_PASSED : QA_FAILED));
	system("pause");
}

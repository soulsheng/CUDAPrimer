#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application, doesn't use cutil library.
*/

#include <stdio.h>
#include <string.h>
#include <iostream>
using namespace std;

#include <hip/hip_runtime.h>

#include <cutil.h>
#include <cutil_inline_runtime.h>   

#define DATA_SIZE (1<<26)//1048576
#define THREAD_NUM  (1<<6)//64
#define BLOCK_NUM    (1<<7)//128


int data[DATA_SIZE];

// ��ʱ�����ο���http://soulshengbbs.sinaapp.com/thread-12-1-1.html ��cuda����ʱ��ķ������ܡ�����cutGetTimerValue
unsigned int hTimer ;
void timeBegin()
{
	cutilDeviceSynchronize() ;
	cutStartTimer(hTimer) ;
}
void timeEnd(string msg)
{
	cutilDeviceSynchronize() ;
	cutStopTimer(hTimer) ;

	double Passed_Time = cutGetTimerValue(hTimer);

	printf("time��%s��: %.3f ms\n", msg.c_str(), Passed_Time);
}

bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

	int i;
    for( i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);
	
    printf("CUDA initialized.\n\n");
    return true;
}



void GenerateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}


__global__ static void sumOfSquares(int *num, int* result)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

	int sum = 0;
	int i;

	for(i = tid + bid * THREAD_NUM; i < DATA_SIZE; i+= THREAD_NUM * BLOCK_NUM) {
		sum += num[i] * num[i];
	}

	result[tid + bid * THREAD_NUM] = sum;
}

void runCUDA()
{
#if 1
	//timeBegin();
	int* gpudata, *result;
	hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
	hipMalloc((void**) &result, sizeof(int) * THREAD_NUM * BLOCK_NUM);
	hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);
	//timeEnd("cuda Memcpy Host To Device");
#endif

#if 1
	//timeBegin();
	sumOfSquares<<<BLOCK_NUM, THREAD_NUM, 0>>>(gpudata, result);
	//timeEnd("kernel");
#endif

#if 1
	//timeBegin();
	int sum[THREAD_NUM * BLOCK_NUM];
	hipMemcpy( &sum, result, sizeof(int) * THREAD_NUM * BLOCK_NUM , hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	//timeEnd("cuda Memcpy Device To Host");
#endif

#if 1
	//timeBegin();
	int final_sum = 0;
	for(int i = 0; i < BLOCK_NUM * THREAD_NUM; i++) {
		final_sum += sum[i] ;
	}
	//timeEnd("cpu add further");
#endif
}

void runCPU()
{
	int final_sum = 0;
	 for(int i = 0; i < DATA_SIZE; i++) {
		 final_sum += data[i] * data[i];
	 }
	// printf("sum��CPU��: %d of %d squares\n", final_sum, DATA_SIZE);
}

int main(int argc, char **argv)
{
	// ��ʼ��cuda
	if(!InitCUDA()) {
		return 0;
	}

	// ���ݳ�ʼ��
	GenerateNumbers(data, DATA_SIZE);

	cutCreateTimer(&hTimer);

	timeBegin();
	// cuda����
	runCUDA();
	timeEnd("CUDA");
	


	// cpu����
	runCPU();


	cutDeleteTimer(hTimer);

	system("pause");
}
